
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define WIDTH 15
#define HEIGHT 10
#define BLUR_SIZE 1

// Three channels corresponding to RGB
struct Pixel {
	unsigned char r;
	unsigned char g;
	unsigned char b;
};

	__global__ 
void imageBlurKernel(struct Pixel * d_out, struct Pixel * d_in, int w, int h)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	if (col < w && row < h) {
		int sumPixRed = 0;
		int sumPixGreen = 0;
		int sumPixBlue = 0;
		int numPix = 0;

		// Get the average of the surrounding BLUR_SIZE x BLUR_SIZE box
		for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow) {
			for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol) {
				int curRow = row + blurRow;
				int curCol = col + blurCol;

				// Verify we have a valid image pixel
				if (curRow >= 0 && curRow <= h - 1 && curCol >= 0 && curCol <= w - 1) {
					// Keep track of number of pixels in the average
					sumPixRed += d_in[curRow * w + curCol].r;
					sumPixGreen += d_in[curRow * w + curCol].g;
					sumPixBlue += d_in[curRow * w + curCol].b;
					numPix++;
				}
			}
		}

		// Write our new pixel value out
		d_out[row * w + col].r = (unsigned char) round(sumPixRed / (float) numPix);
		d_out[row * w + col].g = (unsigned char) round(sumPixGreen / (float) numPix);
		d_out[row * w + col].b = (unsigned char) round(sumPixBlue / (float) numPix);
	}
}

void imageBlur(struct Pixel *h_out, struct Pixel *h_in, int w, int h)
{
	int size = h * w * sizeof(struct Pixel);

	struct Pixel *d_in, *d_out;
	hipMalloc((void **) &d_in, size);
	hipMalloc((void **) &d_out, size);

	hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

	dim3 dimGrid(ceil(w / (float) 8), ceil(h / (float) 8), 1);
	dim3 dimBlock(8, 8, 1);
	imageBlurKernel<<<dimGrid,dimBlock>>>(d_out, d_in, w, h);

	hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);
}

int main(int argc, char *argv[])
{
	int size = HEIGHT * WIDTH * sizeof(struct Pixel);

	struct Pixel *h_in = (struct Pixel *) malloc(size);
	struct Pixel *h_out = (struct Pixel *) malloc(size);

	printf("Clear image:\n");
	for(int i = 0; i < HEIGHT; i++) {
		for(int j = 0; j < WIDTH; j++) {
			// Randomly generate a color image
			h_in[i * WIDTH + j].r = rand() % 256;
			h_in[i * WIDTH + j].g = rand() % 256;
			h_in[i * WIDTH + j].b = rand() % 256;
			printf("Input pixel (%d, %d) = (%d, %d, %d)\n", i, j, 
					h_in[i * WIDTH + j].r, 
					h_in[i * WIDTH + j].g, 
					h_in[i * WIDTH + j].b);
		}
	}

	imageBlur(h_out, h_in, WIDTH, HEIGHT);

	printf("Blur image:\n");
	for(int i = 0; i < HEIGHT; i++) {
		for(int j = 0; j < WIDTH; j++) {
			printf("Output pixel (%d, %d) = (%d, %d, %d)\n", i, j, 
					h_out[i * WIDTH + j].r, 
					h_out[i * WIDTH + j].g, 
					h_out[i * WIDTH + j].b);
		}
	}

	free(h_in);
	free(h_out);
	return 0;
}

