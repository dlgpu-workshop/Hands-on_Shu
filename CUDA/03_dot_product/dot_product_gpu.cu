
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

// Each thread performs one pair-wise addition
__global__ void dotProdKernel(float* d_A, float* d_B, float* d_C, int n)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n)
		d_C[i] = d_A[i] * d_B[i];
}

void dotProduct(float* h_A, float* h_B, float* h_C, int n)
{
	int size = n * sizeof(float);
	float *d_A, *d_B, *d_C;    // pointers to device copies of A, B, C

	// Allocate device memory space for device copies of A, B, C
	hipMalloc((void **) &d_A, size);
	hipMalloc((void **) &d_B, size);
	hipMalloc((void **) &d_C, size);

	// Copy vectors A and B from host memory to device memory
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	// Launch the kernel function to have the device to perform the actual vector addition
	int threads_per_block = 32;
	int no_of_blocks = ceil(n / (float) threads_per_block);	
	dotProdKernel<<<no_of_blocks, threads_per_block>>>(d_A, d_B, d_C, n);

	// Copy result vector C from the device memory to host memory
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	// Free device memory for A, B, C
	hipFree(d_A); 
	hipFree(d_B); 
	hipFree(d_C);
	
	for (int i = 1; i < n; i++)
		h_C[0] += h_C[i];
}

int main(int argc, char *argv[]) {
	float *h_A, *h_B, *h_C;
	int n = 64;
	int size = n * sizeof(float);

	// Memory allocation for h_A, h_B and h_C
	h_A = (float *) malloc(size);
	h_B = (float *) malloc(size);
	h_C = (float *) malloc(size);

	// Setup input values into each of n elements of h_A and h_B
	printf("A = [");
	for (int i = 0; i < n; i++) {
		h_A[i] = (float) i;
		printf(" %.1f ", h_A[i]);
	}
	printf("]\nB = [");
	for (int i = 0; i < n; i++) {
		h_B[i] = (float) i;
		printf(" %.1f ", h_B[i]);
	}
	printf("]\n");

	dotProduct(h_A, h_B, h_C, n);    // Call the host function for dot product

	// Output the results
	printf("Dot Product = %.1f\n", h_C[0]);

	// Free host memory for A and B
	free(h_A);
	free(h_B);
	return 0;
}
